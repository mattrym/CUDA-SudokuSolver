#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "sudoku.h"
#include "gpu_solver.h"

inline void check_cuda_error(hipError_t cuda_status, const char* file, int line)
{
	if (cuda_status != hipSuccess)
	{
		fprintf(stderr, "CUDA error (%s:%d): %s", file, line, hipGetErrorString(cuda_status));
		exit(EXIT_FAILURE);
	}
}
#define CUDA_SAFE(cuda_status) check_cuda_error(cuda_status, __FILE__, __LINE__)

__device__ int cell_index(const int cell)
{
	int row, col;
	int mask_type, mask_offset;

	mask_type = threadIdx.x / N;
	mask_offset = threadIdx.x % N;

	switch (mask_type)
	{
	case ROW_MASK:
		row = mask_offset;
		col = cell;
		break;
	case COL_MASK:
		row = cell;
		col = mask_offset;
		break;
	case SUB_MASK:
		row = (mask_offset / n) * n + cell / n;
		col = (mask_offset % n) * n + cell % n;
		break;
	}

	return row * N + col;
}

__device__ void calculate_masks(const BOARD board, MASK* masks, BLOCK_FLAGS* flags)
{
	int cell, cell_i, value, value_mask;

	masks[threadIdx.x] = 0;

	for (cell = 0; cell < N; ++cell)
	{
		cell_i = cell_index(cell);
		if (value = board[cell_i])
		{
			if ((value_mask = 1 << (value - 1)) & masks[threadIdx.x])
			{
				flags->error = 1;
				break;
			}
			masks[threadIdx.x] |= value_mask;
		}
	}
}

__device__ void find_candidates(BOARD board, MASK* masks, CANDIDATES* candidates, BLOCK_FLAGS* flags)
{
	int row, sub;
	int col, lcol, ucol;
	int cell, digit;
	MASK row_mask, col_mask, sub_mask;

	row = threadIdx.x / n;
	sub = (threadIdx.x / N) * n + threadIdx.x % n;

	row_mask = masks[ROW_MASK * N + row];
	sub_mask = masks[SUB_MASK * N + sub];

	lcol = (threadIdx.x % n) * n;
	ucol = lcol + n;

	for (col = lcol; col < ucol; ++col)
	{
		col_mask = masks[COL_MASK * N + col];
		cell = row * N + col;
		candidates[cell] = row_mask | col_mask | sub_mask;

		if (!board[cell])
		{
			for (digit = 0; digit < N; ++digit)
			{
				if (!(candidates[cell] ^ (1 << digit) ^ FULL_MASK))
				{
					board[cell] = (digit + 1);
					flags->progress = 1;
					break;
				}
			}
			if (!board[cell])
			{
				flags->success = 0;
			}
		}
	}
}

__device__ void find_fork_cell(BOARD board, CANDIDATES* candidates, int* min_forks, int* fork_cell)
{
	int digit, cell, offset;
	int forks[n];

	offset = threadIdx.x * n;
	for (cell = 0; cell < n; ++cell)
	{
		forks[cell] = N;
		if (!board[offset + cell])
		{
			for (digit = 0; digit < N; ++digit)
			{
				forks[cell] -= ((candidates[offset + cell] >> digit) & 1);
			}
		}

		__iAtomicMin(min_forks, forks[cell]);
	}
	__syncthreads();

	for (cell = 0; cell < n; ++cell)
	{
		if (forks[cell] == *min_forks)
		{
			__iAtomicMin(fork_cell, offset + cell);
		}
	}
}

__device__ void fork_board(BOARD boards, int* block_flags, BOARD board, CANDIDATES candidates, int fork_cell)
{
	int digit, forks, block;
	int block_id;
	BOARD forked_board;

	forks = 0;
	block = 0;

	for (digit = 0; digit < N; ++digit)
	{
		if (!((candidates >> digit) & 1))
		{
			if (forks++)
			{
				for (; block < gridDim.x; ++block)
				{
					block_id = gridDim.x * blockIdx.x + fork_cell + SUCCESS;
					__iAtomicCAS(&block_flags[block], IDLE, block_id);

					if (block_flags[block] == block_id)
					{
						forked_board = boards + block * BOARD_SIZE;

						memcpy(forked_board, board, BOARD_SIZE);
						forked_board[fork_cell] = digit + 1;
						block_flags[block] = BUSY;

						break;
					}
				}

			}
			else
			{
				forked_board = boards + blockIdx.x * BOARD_SIZE;

				memcpy(forked_board, board, BOARD_SIZE);
				forked_board[fork_cell] = digit + 1;
			}
		}
	}
}

__global__ void solve_kernel(BOARDS boards, BLOCK_STATUS* block_status)
{
	__shared__ MASK masks[N * n];
	__shared__ CANDIDATES candidates[BOARD_SIZE];
	__shared__ BLOCK_FLAGS flags;

	__shared__ int min_forks;
	__shared__ int fork_cell;

	__shared__ CELL board[BOARD_SIZE];
	BOARD last_board;

	if (block_status[blockIdx.x] != BUSY)
	{
		return;
	}

	if (!threadIdx.x)
	{
		min_forks = 9;
		fork_cell = BOARD_SIZE;

		memcpy(board, boards + blockIdx.x * BOARD_SIZE, BOARD_SIZE);
		last_board = boards + BLOCKS * BOARD_SIZE;
	}
	__syncthreads();

	do
	{
		flags = { 0, 1, 0 };

		calculate_masks(board, masks, &flags);
		__syncthreads();

		find_candidates(board, masks, candidates, &flags);
		__syncthreads();
	} while (!flags.error && !flags.success && flags.progress);

	if (flags.error)
	{
		if (!threadIdx.x)
		{
			block_status[blockIdx.x] = IDLE;
		}
		return;
	}

	if (flags.success)
	{
		if (!threadIdx.x)
		{
			block_status[BLOCKS] = SUCCESS;
			memcpy(last_board, board, BOARD_SIZE * sizeof(CELL));
		}
		return;
	}

	find_fork_cell(board, candidates, &min_forks, &fork_cell);
	__syncthreads();

	if (!threadIdx.x)
	{
		fork_board(boards, block_status, board, candidates[fork_cell], fork_cell);
	}
}

void solve_gpu(const BOARD input_board, BOARD output_board)
{
	BOARDS boards;
	BLOCK_STATUS* block_status;
	BLOCK_STATUS last_block_status;

	int it;

	CUDA_SAFE(hipSetDevice(0));

	CUDA_SAFE(hipMalloc((void**)&block_status, (BLOCKS + 1) * sizeof(BLOCK_STATUS)));
	CUDA_SAFE(hipMemset(block_status, 1, 1));

	CUDA_SAFE(hipMalloc((void**)&boards, (BLOCKS + 1) * BOARD_SIZE * sizeof(CELL)));
	CUDA_SAFE(hipMemcpy(boards, input_board, BOARD_SIZE * sizeof(CELL), hipMemcpyHostToDevice));

	for (it = 0; it < ITERATIONS; ++it)
	{
		solve_kernel<<<BLOCKS, THREADS>>>(boards, block_status);
		CUDA_SAFE(hipGetLastError());
		CUDA_SAFE(hipDeviceSynchronize());

		CUDA_SAFE(hipMemcpy(&last_block_status, block_status + BLOCKS, sizeof(BLOCK_STATUS), hipMemcpyDeviceToHost));
		if (last_block_status == SUCCESS)
		{
			CUDA_SAFE(hipMemcpy(output_board, boards + BLOCKS * BOARD_SIZE, BOARD_SIZE * sizeof(CELL), hipMemcpyDeviceToHost));
			break;
		}
	}

	hipFree(boards);
	hipFree(block_status);

	CUDA_SAFE(hipDeviceReset());
}